#include "hip/hip_runtime.h"
#include "cpu_anim.h"
#define DIM 1024
#define PI 3.1415926535897932f
struct DataBlock
{
	CPUAnimBitmap *bitmap;
	unsigned char * dev_bitmap;
};
__global__ void kernel(unsigned char *ptr,int ticks)
{
	int x=threadIdx.x + blockIdx.x*blockDim.x;
	int y=threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y*blockDim.x*gridDim.x;

	float fx = x - DIM/2;
	float fy = y - DIM/2;
	float d = sqrtf(fx*fx + fy*fy);

	unsigned char grey = (unsigned char)(128.0f + 127.0f *cos(d/10.0f - ticks/7.0f) /(d/10.0f + 1.0f));

	ptr[offset*4+0]=grey;
	ptr[offset*4+1]=grey;
	ptr[offset*4+2]=grey;
	ptr[offset*4+3]=255;
}
void generate_frame(DataBlock *d,int ticks)
{
	dim3 grids(DIM/16,DIM/16);
	dim3 threads(16,16);
	kernel<<<grids,threads>>>(d->dev_bitmap,ticks);
	hipMemcpy(d->bitmap->get_ptr(),d->dev_bitmap,d->bitmap->image_size(),hipMemcpyDeviceToHost);
}
void clean_up(DataBlock *d)
{
	hipFree(d->dev_bitmap);
	delete d->bitmap;
}
int main()
{
	DataBlock data;
	data.bitmap = new CPUAnimBitmap(DIM,DIM,&data);
	hipMalloc((void **)&data.dev_bitmap,data.bitmap->image_size());
	data.bitmap->anim_and_exit((void (*)(void *,int))generate_frame,(void (*)(void *))clean_up);
}
