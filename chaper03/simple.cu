
#include <hip/hip_runtime.h>
#include<iostream>
__global__ void add(int a,int b,int *c)
{
	 *c = a+b;
}
int main()
{
	int c;
	int *dev_c;
	hipMalloc((void **)&dev_c,sizeof(int));
	add<<<1,1>>>(2,7,dev_c);
	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
	std::cout<<c<<std::endl;
	hipFree(dev_c);
	return 0;
}
